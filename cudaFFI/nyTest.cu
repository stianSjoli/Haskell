
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C"
int example(int a, int b){
	return a + b;
}

extern "C"
void example2(int input){
	printf("%s%d\n", "hello ", input);
}

extern "C"
void gpuTest(void){
	int n = 10;
	int a[n], b[n], c[n];
	int *dev_a, *dev_b, *dev_c;

	hipMalloc( (void**)&dev_a, n * sizeof(int) ); 
	hipMalloc( (void**)&dev_b, n * sizeof(int) ); 
	hipMalloc( (void**)&dev_c, n * sizeof(int) );

	for (int i=0; i<n; i++) { 
		a[i] = -i;
		b[i] = i * i; 
	}

	hipMemcpy( dev_a, a, n * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, n * sizeof(int), hipMemcpyHostToDevice );
    //add<<<n,1>>>( dev_a, dev_b, dev_c );
       // copy the array 'c' back from the GPU to the CPU
	hipMemcpy( c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost );
       // display the results
	for (int i=0; i<n; i++) {
		printf( "%d + %d = %d\n", a[i], b[i], c[i] );
	}
       // free the memory allocated on the GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
}
/*
int main(){
	gpuTest();
	printf("hello");
}
*/